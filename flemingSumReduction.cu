#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
//Code written by Alan Fleming

//CONSTANTS
#define MATRIXSIZE 131072
#define BLOCKSIZE 1024


//Code to preform sum reduction using the cpu
int SumReductionCPU(int* x, int N){
	int sum = 0;
	for(int i = 0; i < N; i++){
		sum += x[i];
	}
	return sum;
}

__global__ void sumReductionKernal(int* arr) {

	//initialize Partial Result for thread	
	__shared__ int partialResult[2 * BLOCKSIZE];
	unsigned int start = 2*blockIdx.x * blockDim.x;
	partialResult[threadIdx.x] = arr[start + threadIdx.x];
	partialResult[blockDim.x + threadIdx.x] = arr[start +blockDim.x + threadIdx.x];
	
	//Preform sum reduction
	for(unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
		__syncthreads();
		if (threadIdx.x < stride){
			partialResult[threadIdx.x] += partialResult[threadIdx.x + stride];
		}
	}
	
	__syncthreads();
	if(threadIdx.x == 0){
		//write block sum to global memory
		arr[blockIdx.x] = partialResult[0];
	}
}

int main() {
	
	int *a = (int *)malloc(sizeof(int) * MATRIXSIZE); //allocate space for array
	//initialize array
	int init = 1325;
	for(int i=0; i<MATRIXSIZE;i++){
		init = 3125 * init % 6553;
		a[i] = (init - 1000) % 97;
	}

	//Test CPU reduction
	//Get start time
	clock_t t1 = clock();
	//Calculate reduction
	int cpuResult = SumReductionCPU(a, MATRIXSIZE);
	//Get stop time
	clock_t t2 = clock();
	//Calculate runtime
	float cpuTime= (float(t2-t1)/CLOCKS_PER_SEC*1000);

	//Allocate memory on GPU compution
	int *dev_a;
	hipMalloc((void **)(&dev_a), MATRIXSIZE *sizeof(int));

	//copy memory to gpu
	hipMemcpy(dev_a,a, MATRIXSIZE * sizeof(int), hipMemcpyHostToDevice);

	//calculate dimentions for gpu
	dim3 dimBlock(BLOCKSIZE);
	dim3 dimGrid(ceil(double(MATRIXSIZE)/dimBlock.x));

	//Set up cuda events for recording runtime
	hipEvent_t start,stop;
	float gpuTime; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	//Calculate GPU Reduction for each block
	sumReductionKernal<<<dimGrid, dimBlock>>>(dev_a);
	//Calculate GPU Recuction for block results
	sumReductionKernal<<<dimGrid, dimBlock>>>(dev_a);

	//calculate runtime 
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime,start,stop);

	//destroy cuda events
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//copy sum from gpu
	hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

	//print speedup
	printf("CPU Runtime: %f\nGpu Runtime: %f\nSpeedup: %f\n", (double)cpuTime, (double)gpuTime, double(cpuTime / gpuTime));

	//print reduction results
	printf("CPU Result: %d\nGPU Result: %d\n", cpuResult, a[0]);
	//verify results
	if(cpuResult == a[0]) {
		printf("TEST PASSED\n");
	} else {
		printf("TEST FAILED\n");
	}

	//free memory
	free(a);
	hipFree(dev_a);
	
	return 0;
}
